#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include <iostream>
#include <sstream>
#include "world_priv.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#define CUDA_SM		 128
#define CUDA_WARP_SIZE	 32
#define CUDA_BLOCK_DIM_X 32
#define CUDA_BLOCK_DIM_Y 32
#define CUDA_NB_THREAD	 (CUDA_BLOCK_DIM_X * CUDA_BLOCK_DIM_Y)
#define CUDA_NB_BLOCK	 (CUDA_SM / CUDA_WARP_SIZE)

#define FatalError(s)                                                          \
	do {                                                                   \
		std::cout << std::flush << "ERROR: " << s << " in "            \
			  << __FILE__ << ':' << __LINE__ << "\nAborting...\n"; \
		hipDeviceReset();                                             \
		exit(-1);                                                      \
	} while (0)

#define checkCudaErrors(status)                                                \
	do {                                                                   \
		std::stringstream _err;                                        \
		if (status != 0) {                                             \
			_err << "cuda failure (" << hipGetErrorString(status) \
			     << ')';                                           \
			FatalError(_err.str());                                \
		}                                                              \
	} while (0)

static __global__ void world_init_random_generator(hiprandState *state,
						   size_t len, uint64_t seed)
{
	const size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t j = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t index = i * gridDim.x * blockDim.x + j;
	if (index < len) {
		hiprand_init(seed, index, 0, &state[index]);
	}
}

static inline __device__ bool should_happen(int probability, hiprandState *state)
{
	double rand_value = hiprand_uniform(state);
	return rand_value < ((double)probability / 100);
}

static __device__ uint8_t world_get_nb_infected_neighbours(const world_t *p,
							   size_t i, size_t j)
{
	uint8_t sum = 0;
	for (int dx = -p->params.proximity; dx <= p->params.proximity; ++dx) {
		for (int dy = -p->params.proximity; dy <= p->params.proximity;
		     ++dy) {
			if (dx == 0 && dy == 0) {
				continue;
			}
			const int ni = i + dx;
			const int nj = j + dy;
			if (!(ni < p->params.worldHeight &&
			      nj < p->params.worldWidth)) {
				continue;
			}

			sum += p->grid[ni * p->params.worldWidth + nj] ==
			       INFECTED;
		}
	}
	return sum;
}

typedef struct {
	world_t *d_world;
	state_t *d_curr_grid;
	// store this so we can free them later
	state_t *d_tmp_grid;
	uint8_t *d_infection_duration_grid;
} cuda_prepare_update_t;

static cuda_prepare_update_t cuda_prepare;

static __global__ void init_population_kernel(
	state_t *grid, const world_parameters_t *p, size_t people_to_spawn,
	hiprandState *random_states,
	int *occupation_buffer) // Buffer used to lock a random position
{
	const size_t world_size = p->worldWidth * p->worldHeight;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= p->worldHeight || j >= p->worldWidth) {
		return;
	}

	int index = i * p->worldWidth + j;
	if (index >= people_to_spawn) {
		return;
	}

	state_t state;
	if (index < p->initialInfected) {
		state = INFECTED;
	} else if (index < p->initialInfected + p->initialImmune) {
		state = IMMUNE;
	} else {
		state = HEALTHY;
	}

	bool found_position = false;
	while (!found_position) {
		size_t pos = hiprand(&random_states[index]) % world_size;
		// if occupation == 0 then write 1 to define cell usage
		if (atomicCAS(&occupation_buffer[pos], 0, 1) == 0) {
			grid[pos] = state;
			found_position = true;
		}
	}
}

int world_init(world_t *world, const world_parameters_t *p)
{
	int err = world_init_common(world, p);
	if (err < 0) {
		return err;
	}

	hiprandState *d_state;
	const size_t world_size = world_world_size(p);
	dim3 block(CUDA_BLOCK_DIM_X, CUDA_BLOCK_DIM_Y);
	dim3 grid((p->worldWidth + CUDA_BLOCK_DIM_X - 1) / CUDA_BLOCK_DIM_X,
		  (p->worldHeight + CUDA_BLOCK_DIM_Y - 1) / CUDA_BLOCK_DIM_Y);
	checkCudaErrors(
		hipMalloc((void **)&d_state, world_size * sizeof(*d_state)));

	world_init_random_generator<<<grid, block>>>(d_state, world_size,
						       1337);
	checkCudaErrors(hipDeviceSynchronize());

	world->cuda_random_state = (void *)d_state;

	state_t *d_grid;

	hipMalloc(&d_grid, world_size * sizeof(state_t));

	hipMemset(d_grid, EMPTY, world_size * sizeof(state_t));

	int *d_occupation_buffer;
	hipMalloc((void **)&d_occupation_buffer,
		   world_size * sizeof(*d_occupation_buffer));

	hipMemset(d_occupation_buffer, 0,
		   world_size * sizeof(*d_occupation_buffer));
	world_parameters_t *d_p;

	hipMalloc((void **)&d_p, sizeof(*d_p));
	hipMemcpy(d_p, p, sizeof(*d_p), hipMemcpyHostToDevice);

	const size_t people_to_spawn = world_initial_population(p);

	init_population_kernel<<<grid, block>>>(d_grid, d_p, people_to_spawn,
						  d_state, d_occupation_buffer);

	checkCudaErrors(hipDeviceSynchronize());

	hipMemcpy(world->grid, d_grid, world_size * sizeof(state_t),
		   hipMemcpyDeviceToHost);

	hipFree(d_grid);

	return 0;
}

bool __device__ world_should_infect(world_t *p, size_t i, size_t j,
				    int probability)
{
	return world_get_nb_infected_neighbours(p, i, j) &&
	       should_happen(probability,
			     &((hiprandState *)p->cuda_random_state)
				     [i * p->params.worldWidth + j]);
}
void __device__ world_infect_if_should_infect(world_t *p, state_t *grid,
					      size_t i, size_t j,
					      int probability)
{
	if (world_should_infect(p, i, j, probability)) {
		const size_t index = i * p->params.worldWidth + j;
		grid[index] = INFECTED;
		p->infectionDurationGrid[index] = p->params.infectionDuration;
	}
}
void __device__ world_handle_infected(world_t *p, state_t *world, size_t i,
				      size_t j)
{
	const size_t index = i * p->params.worldWidth + j;

	if (p->infectionDurationGrid[index] == 0) {
		if (should_happen(
			    p->params.deathProbability,
			    &((hiprandState *)p->cuda_random_state)[index])) {
			world[index] = DEAD;
		} else {
			world[index] = IMMUNE;
			p->infectionDurationGrid[index] =
				p->params.infectionDuration;
		}
	} else {
		p->infectionDurationGrid[index]--;
	}
}

static __global__ void world_update_k(world_t *w, state_t *result_grid)
{
	size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	size_t j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < w->params.worldHeight && j < w->params.worldWidth) {
		size_t index = i * w->params.worldWidth + j;

		result_grid[index] = w->grid[index];

		switch (w->grid[index]) {
		case HEALTHY:
			world_infect_if_should_infect(
				w, result_grid, i, j,
				w->params.healthyInfectionProbability);
			break;
		case IMMUNE:
			world_infect_if_should_infect(
				w, result_grid, i, j,
				w->params.immuneInfectionProbability);
			break;
		case INFECTED:
			world_handle_infected(w, result_grid, i, j);
			break;
		case EMPTY:
		case DEAD:
			break;
		}
	}
}
void world_update(world_t *p, void *raw)
{
	(void)raw;
	const size_t world_size = world_world_size(&p->params);
	const size_t GRID_SIZE = world_size * sizeof(state_t);
	const size_t INFECTION_GRID_SIZE = world_size * sizeof(uint8_t);
	state_t *d_grid;
	state_t *d_tmp_grid;
	uint8_t *d_infection_duration_grid;

	checkCudaErrors(hipMalloc((void **)&d_grid, GRID_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_tmp_grid, GRID_SIZE));
	checkCudaErrors(hipMalloc((void **)&d_infection_duration_grid,
				   INFECTION_GRID_SIZE));

	world_t world;

	world.grid = d_grid;
	world.infectionDurationGrid = d_infection_duration_grid;
	world.params = p->params;
	world.cuda_random_state = p->cuda_random_state;

	world_t *d_world;

	checkCudaErrors(hipMalloc((void **)&d_world, sizeof(world_t)));

	checkCudaErrors(
		hipMemcpy(d_grid, p->grid, GRID_SIZE, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_tmp_grid, p->grid, GRID_SIZE,
				   hipMemcpyHostToDevice));
	checkCudaErrors(
		hipMemcpy(d_infection_duration_grid, p->infectionDurationGrid,
			   INFECTION_GRID_SIZE, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_world, &world, sizeof(world_t),
				   hipMemcpyHostToDevice));

	cuda_prepare.d_world = d_world;
	cuda_prepare.d_curr_grid = d_grid;
	cuda_prepare.d_tmp_grid = d_tmp_grid;
	cuda_prepare.d_infection_duration_grid = d_infection_duration_grid;

	size_t infected_before = world_get_infected(p);

	dim3 block(CUDA_BLOCK_DIM_X, CUDA_BLOCK_DIM_Y);
	dim3 grid((p->params.worldWidth + block.x - 1) / block.x,
		  (p->params.worldHeight + block.y - 1) / block.y);
	world_update_k<<<grid, block>>>(cuda_prepare.d_world,
					  cuda_prepare.d_tmp_grid);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipMemcpy(p->grid, cuda_prepare.d_tmp_grid, GRID_SIZE,
				   hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(p->infectionDurationGrid,
				   cuda_prepare.d_infection_duration_grid,
				   INFECTION_GRID_SIZE,
				   hipMemcpyDeviceToHost));

	size_t infected_after = world_get_infected(p);

	hipFree(cuda_prepare.d_tmp_grid);
	hipFree(cuda_prepare.d_curr_grid);
	hipFree(cuda_prepare.d_infection_duration_grid);
	hipFree(cuda_prepare.d_world);
}
void *world_prepare_update(const world_t *p)
{
	return (void *)&cuda_prepare;
}

void world_destroy(world_t *w)
{
	hipFree(w->cuda_random_state);
	world_destroy_common(w);
}

#endif
