#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include "world_priv.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#define CUDA_SM		 128
#define CUDA_WARP_SIZE	 32
#define CUDA_BLOCK_DIM_X 16
#define CUDA_BLOCK_DIM_Y 16
#define CUDA_NB_THREAD	 (CUDA_BLOCK_DIM_X * CUDA_BLOCK_DIM_Y)
#define CUDA_NB_BLOCK	 (CUDA_SM / CUDA_WARP_SIZE)

static __global__ void world_init_random_generator(hiprandState *state,
						uint64_t seed)
{
	const size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t j = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t index = i * gridDim.x * blockDim.x + j;
	hiprand_init(seed, index, 0, &state[index]);
}

static inline __device__ bool should_happen(int probability, hiprandState *state)
{
	double rand_value = hiprand_uniform(state);
	return rand_value < ((double)probability / 100);
}

static __device__ uint8_t world_get_nb_infected_neighbours(const world_t *p,
							   size_t i, size_t j)
{
	uint8_t sum = 0;
	for (int dx = -p->params.proximity; dx <= p->params.proximity; ++dx) {
		for (int dy = -p->params.proximity; dy <= p->params.proximity;
		     ++dy) {
			if (dx == 0 && dy == 0) {
				continue;
			}
			const size_t ni = i + dx;
			const size_t nj = j + dy;
			if (!(ni < p->params.worldHeight &&
			      nj < p->params.worldWidth)) {
				continue;
			}

			sum += p->grid[ni * p->params.worldWidth + nj] ==
			       INFECTED;
		}
	}
	return sum;
}

typedef struct {
	world_t *d_world;
	state_t *d_curr_grid;
	// store this so we can free them later
	state_t *d_tmp_grid;
	uint8_t *d_infection_duration_grid;
} cuda_prepare_update_t;

static cuda_prepare_update_t cuda_prepare;

int world_init(world_t *world, const world_parameters_t *p)
{
	int err = world_init_common(world, p);
	if (err < 0) {
		return err;
	}
	hiprandState *d_state;
	hipMalloc((void **)&d_state,
		   CUDA_NB_THREAD * CUDA_NB_BLOCK * sizeof(*d_state));
	dim3 block(CUDA_BLOCK_DIM_X, CUDA_BLOCK_DIM_Y);
	dim3 grid((p->worldWidth + block.x - 1) / block.x,
		  (p->worldHeight + block.y - 1) / block.y);

	world_init_random_generator<<<grid, block>>>(d_state, 1337);

	/* No need to synchronize here */
	world->random_state = d_state;

	return 0;
}

bool __device__ world_should_infect(world_t *p, size_t i, size_t j,
				    int probability)
{
	return world_get_nb_infected_neighbours(p, i, j) &&
	       should_happen(probability,
			     &p->random_state[i * p->params.worldWidth + j]);
}
void __device__ world_infect_if_should_infect(world_t *p, state_t *grid,
					      size_t i, size_t j,
					      int probability)
{
	if (world_should_infect(p, i, j, probability)) {
		grid[i * p->params.worldWidth + j] = INFECTED;
	}
}
void __device__ world_handle_infected(world_t *p, state_t *world, size_t i,
				      size_t j)
{
	const size_t index = i * p->params.worldWidth + j;

	if (p->infectionDurationGrid[index] == 0) {
		if (should_happen(p->params.deathProbability,
				  &p->random_state[index])) {
			world[index] = DEAD;
		} else {
			world[index] = IMMUNE;
			p->infectionDurationGrid[index] =
				p->params.infectionDuration;
		}
	} else {
		p->infectionDurationGrid[index]--;
	}
}

static __global__ void world_update_k(world_t *w, state_t *result_grid)
{
	size_t i = blockIdx.y * blockDim.y + threadIdx.y;
	size_t j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < w->params.worldHeight && j < w->params.worldWidth) {
		int index = i * w->params.worldWidth + j;
		switch (w->grid[index]) {
		case HEALTHY:
			world_infect_if_should_infect(
				w, result_grid, i, j,
				w->params.healthyInfectionProbability);
			break;
		case IMMUNE:
			world_infect_if_should_infect(
				w, result_grid, i, j,
				w->params.immuneInfectionProbability);
			break;
		case INFECTED:
			world_handle_infected(w, result_grid, i, j);
			break;
		case EMPTY:
		case DEAD:
			break;
		}
	}
	size_t index = i * w->params.worldWidth + j;
	w->grid[index] = result_grid[index];
}
void world_update(world_t *p, void *raw)
{
	cuda_prepare_update_t *update_data = (cuda_prepare_update_t *)raw;
	const size_t world_size = world_world_size(&p->params);

	dim3 block(CUDA_BLOCK_DIM_X, CUDA_BLOCK_DIM_Y);
	dim3 grid((p->params.worldWidth + blockDim.x - 1) / blockDim.x,
		  (p->params.worldHeight + blockDim.y - 1) / blockDim.y);
	world_update_k<<<grid, block>>>(update_data->d_world,
					  update_data->d_tmp_grid);

	hipMemcpy(p->grid, update_data->d_tmp_grid,
		   world_size * sizeof(*p->grid), hipMemcpyDeviceToHost);
	hipMemcpy(p->infectionDurationGrid,
		   update_data->d_infection_duration_grid,
		   world_size * sizeof(*p->infectionDurationGrid),
		   hipMemcpyDeviceToHost);

	hipFree(update_data->d_tmp_grid);
	hipFree(update_data->d_curr_grid);
	hipFree(update_data->d_infection_duration_grid);
	hipFree(update_data->d_world);
}
void *world_prepare_update(const world_t *p)
{
	const size_t world_size = world_world_size(&p->params);
	int err;
	state_t *d_grid;
	err = hipMalloc((void **)&(d_grid), world_size * sizeof(*d_grid));
	if (err != hipSuccess) {
		return NULL;
	}
	state_t *d_tmp_grid;
	err = hipMalloc((void **)&(d_tmp_grid),
			 world_size * sizeof(*d_tmp_grid));

	if (err != hipSuccess) {
		hipFree(d_grid);
		return NULL;
	}

	uint8_t *d_infection_duration_grid;

	err = hipMalloc((void **)&(d_infection_duration_grid),
			 world_size * sizeof(*d_infection_duration_grid));

	if (err != hipSuccess) {
		hipFree(d_grid);
		hipFree(d_tmp_grid);
		return NULL;
	}
	world_t world;

	world.grid = d_grid;
	world.infectionDurationGrid = d_infection_duration_grid;
	world.params = p->params;

	world_t *d_world;

	err = hipMalloc((void **)&(d_world), sizeof(*d_world));

	if (err != hipSuccess) {
		hipFree(d_grid);
		hipFree(d_tmp_grid);
		hipFree(d_infection_duration_grid);
		return NULL;
	}

	hipMemcpy(d_tmp_grid, p->grid, world_size * sizeof(*d_tmp_grid),
		   hipMemcpyHostToDevice);
	hipMemcpy(d_grid, p->grid, world_size * sizeof(*d_grid),
		   hipMemcpyHostToDevice);
	hipMemcpy(d_infection_duration_grid, p->infectionDurationGrid,
		   world_size * sizeof(*d_infection_duration_grid),
		   hipMemcpyHostToDevice);

	hipMemcpy(d_world, &world, sizeof(world_t), hipMemcpyHostToDevice);

	cuda_prepare.d_world = d_world;
	cuda_prepare.d_curr_grid = d_grid;
	cuda_prepare.d_tmp_grid = d_tmp_grid;
	cuda_prepare.d_infection_duration_grid = d_infection_duration_grid;

	return (void *)&cuda_prepare;
}

void world_destroy(world_t *w)
{
	hipFree(w->random_state);
	world_destroy_common(w);
}

#endif
